
#include <hip/hip_runtime.h>
#ifdef __CUDA_ARCH__
#  if __CUDA_ARCH__ != 500
#    error "Passed architecture 50, but got something else."
#  endif
#endif

// Check HOST_DEFINE only for nvcc
#ifndef __CUDA__
#  ifndef HOST_DEFINE
#    error "HOST_DEFINE not defined!"
#  endif
#endif

int main()
{
}
